#include "hip/hip_runtime.h"
__device__ void ensure_appropriate_values(float e_value, float lognormval, float* bvalue){
	if (!isnan(e_value) && !isinf(e_value)) {
		if (e_value<=1) {
			*bvalue = 0;
		}
		else {
			*bvalue = floorf(logf(e_value)/lognormval);
		}
	}

	*bvalue = (float) min((int) *bvalue,  HiCCUPS_W1_MAX_INDX );
}

__device__ void process_masks_lr(int i_start, int i_max_p1, int msize, int t_col, float *c,float *d, int diff,
	float* evalue_d, float* evalue_dist_d, float* evalue_v, float* evalue_dist_v){

	for (int i = i_start; i < i_max_p1; i++) {
		int index = i * msize + t_col;
		if (!isnan(c[index])) {
			*evalue_d -= c[index];
			*evalue_dist_d -= d[abs(i+diff-t_col)];
		}
		for (int j = -1; j < 2; j++) {
			*evalue_v += c[index + j];
			*evalue_dist_v += d[abs(i+diff-t_col-j)];
		}
	}
}

__device__ void process_masks_tb(int j_start, int j_max_p1, int msize, int t_row, float *c,float *d, int diff,
	float* evalue_d, float* evalue_dist_d, float* evalue_h, float* evalue_dist_h){

	for (int j = j_start; j < j_max_p1; j++) {
		int index = t_row * msize + j;
		if (!isnan(c[index])) {
			*evalue_d -= c[index];
			*evalue_dist_d -= d[abs(t_row+diff-j)];
		}
		for (int i = -1; i < 2; i++) {
			*evalue_h += c[(t_row+i) * msize + j];
			*evalue_dist_h += d[abs(t_row+i+diff-j)];
		}
	}
}

extern "C"
__global__ void BasicPeakCallingKernel(float *c, float *expectedbl, float *expecteddonut, float *expectedh,
	float *expectedv, float *observed, float *b_bl, float *b_donut, float *b_h, float *b_v, float *p,
	float *tbl, float *td, float *th, float *tv, float *d, float *kr1, float *kr2, float *bound1, float *bound3)
{
    // 2D Thread ID
	int t_col = threadIdx.x + blockIdx.x * blockDim.x;
	int t_row = threadIdx.y + blockIdx.y * blockDim.y;

    // Evalue is used to store the element of the matrix
    // that is computed by the thread
	float Evalue_bl =  0;
	float Edistvalue_bl = 0;
	float Evalue_donut =  0;
	float Edistvalue_donut = 0;
	float Evalue_h =  0;
	float Edistvalue_h = 0;
	float Evalue_v =  0;
	float Edistvalue_v = 0;
	float e_bl = 0;
	float e_donut = 0;
	float e_h = 0;
	float e_v = 0;
	float o = 0;
	float sbtrkt = 0;
	float bvalue_bl = 0;
	float bvalue_donut = 0;
	float bvalue_h = 0;
	float bvalue_v = 0;
	int wsize =  HiCCUPS_WINDOW;
	int msize =  HiCCUPS_MATRIX_SIZE;
	int pwidth =  HiCCUPS_PEAK_WIDTH;
	int buffer_width =  HiCCUPS_REGION_MARGIN;
	int diff = bound1[0] - bound3[0];
	int diagDist = abs(t_row+diff-t_col);
	int maxIndex = msize-buffer_width;

	wsize = min(wsize, (abs(t_row+diff-t_col)-1)/2);
	if (wsize <= pwidth) {
		wsize = pwidth + 1;
	}
	wsize = min(wsize, buffer_width);

  // only run if within central window (not in data buffer margins)
	if (t_row >= buffer_width && t_row<maxIndex && t_col>= buffer_width && t_col<maxIndex) {

		// calculate initial bottom left box
		for (int i = t_row+1; i <= t_row+wsize; i++) {
			for (int j = t_col-wsize; j < t_col; j++) {
				int index = i * msize + j;
				if (!isnan(c[index])) {
					if (i+diff-j<0) {
						Evalue_bl += c[index];
						Edistvalue_bl += d[abs(i+diff-j)];
					}
				}
			}
		}

		//Subtract off the middle peak
		for (int i = t_row+1; i <= t_row+pwidth; i++) {
			for (int j = t_col-pwidth; j < t_col; j++) {
				int index = i * msize + j;
				if (!isnan(c[index])) {
					if (i+diff-j<0) {
						Evalue_bl -= c[index];
						Edistvalue_bl -= d[abs(i+diff-j)];
					}
				}
			}
		}

		//fix box dimensions
		while (Evalue_bl<16) {
			Evalue_bl =0;
			Edistvalue_bl =0;
			wsize+=1;
			for (int i = t_row+1; i <= t_row+wsize; i++) {
				for (int j = t_col-wsize; j < t_col; j++) {
					int index = i * msize + j;
					if (!isnan(c[index]) && i+diff-j<0) {
						Evalue_bl += c[index];
						Edistvalue_bl += d[abs(i+diff-j)];
						if (i > t_row && i < t_row+pwidth+1 && j > t_col-pwidth-1 && j < t_col) {
							Evalue_bl -= c[index];
							Edistvalue_bl -= d[abs(i+diff-j)];
						}
					}
				}
			}

			if (wsize >= buffer_width) {
				break;
			}
			if (2*wsize>= abs(t_row+diff-t_col)) {
				break;
			}
		}

		// calculate donut
		for (int i = t_row-wsize; i <= t_row+wsize; ++i) {
			for (int j = t_col-wsize; j <= t_col+wsize; ++j) {
				int index = i * msize + j;
				if (!isnan(c[index])) {
					if (i+diff-j<0) {
						Evalue_donut += c[index];
						Edistvalue_donut += d[abs(i+diff-j)];
					}
				}
			}
		}
		//Subtract off the middle peak
		for (int i = t_row-pwidth; i <= t_row+pwidth; ++i) {
			for (int j = t_col-pwidth; j <= t_col+pwidth; ++j) {
				int index = i * msize + j;
				if (!isnan(c[index])) {
					if (i+diff-j<0) {
						Evalue_donut -= c[index];
						Edistvalue_donut -= d[abs(i+diff-j)];
					}
				}
			}
		}

		//Subtract off the cross hairs left side
		process_masks_lr(t_row-wsize, t_row-pwidth, msize, t_col, c, d, diff, &Evalue_donut, &Edistvalue_donut, &Evalue_v, &Edistvalue_v);

		//Subtract off the cross hairs right side
		process_masks_lr(t_row+pwidth+1, t_row+wsize+1, msize, t_col, c, d, diff, &Evalue_donut, &Edistvalue_donut, &Evalue_v, &Edistvalue_v);

		//Subtract off the cross hairs top side
		process_masks_tb(t_col-wsize, t_col-pwidth, msize, t_row, c, d, diff, &Evalue_donut, &Edistvalue_donut, &Evalue_h, &Edistvalue_h);

		//Subtract off the cross hairs bottom side
		process_masks_tb(t_col+pwidth+1, t_col+wsize+1, msize, t_row, c, d, diff, &Evalue_donut, &Edistvalue_donut, &Evalue_h, &Edistvalue_h);


		e_bl = ((Evalue_bl*d[diagDist])/Edistvalue_bl)*kr1[t_row]*kr2[t_col];
		e_donut = ((Evalue_donut*d[diagDist])/Edistvalue_donut)*kr1[t_row]*kr2[t_col];
		e_h = ((Evalue_h*d[diagDist])/Edistvalue_h)*kr1[t_row]*kr2[t_col];
		e_v = ((Evalue_v*d[diagDist])/Edistvalue_v)*kr1[t_row]*kr2[t_col];

		float lognorm = logf(powf(2.0,.33));

		ensure_appropriate_values(e_bl, lognorm, &bvalue_bl);
		ensure_appropriate_values(e_donut, lognorm, &bvalue_donut);
		ensure_appropriate_values(e_h, lognorm, &bvalue_h);
		ensure_appropriate_values(e_v, lognorm, &bvalue_v);

  		// Write the matrix to device memory;
  		// each thread writes one element
		int val_index = t_row * msize + t_col;
		expectedbl[val_index] = e_bl;
		expecteddonut[val_index] = e_donut;
		expectedh[val_index] = e_h;
		expectedv[val_index] = e_v;
		o = roundf(c[val_index]*kr1[t_row]*kr2[t_col]);
		observed[val_index] = o;
		b_bl[val_index] = bvalue_bl;
		b_donut[val_index] = bvalue_donut;
		b_h[val_index] = bvalue_h;
		b_v[val_index] = bvalue_v;
		sbtrkt = fmaxf(tbl[(int) bvalue_bl],td[(int) bvalue_donut]);
		sbtrkt = fmaxf(sbtrkt, th[(int) bvalue_h]);
		sbtrkt = fmaxf(sbtrkt, tv[(int) bvalue_v]);
		p[val_index] = o-sbtrkt;
	}
}